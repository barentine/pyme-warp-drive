
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void convRowGPU_var(float *var, float *rconvdata, float *filter,// const int rowsize,
int halfFilt, const int colsize)
//
// This function takes input data and performs a row convolution. The convolution is stored in a separate
// output array.
//
{
    int k, halfFiltm1 = halfFilt-1;
    int rid = blockIdx.x;// + halfFilt;
    int j = threadIdx.x;// + halfFilt;
    float tempsum = 0;

    volatile __shared__ float rdata_sh[1075]; //should be changed to colsize (PADDED SIZE, or larger)
    __shared__ float filter_sh[12];
    //rdata_sh[j] = data[rid*colsize + j];


    if (j < (halfFilt)){
        rdata_sh[j] = 0;
        rdata_sh[colsize + j + halfFilt] = 0;
        //printf("colsize + halfFilt %d", (colsize + halfFilt));
    }
    rdata_sh[j + halfFilt] = 1/var[rid*colsize + j];
    if (j < (2*halfFilt)) filter_sh[j] = filter[j];

    //if (j > (colsize - 2*halfFilt)){
    //    filter_sh[j] = filter[j]; //trying to use other threads in case the two threads can run async.
    //    printf("%d,", j);
    //}

    __syncthreads();

    //if ((j==colsize-1)&&(rid==5)) printf("colsize + halfFilt %d", (colsize + j));

    for (k = -halfFilt; k <= halfFiltm1; k++){
        tempsum += rdata_sh[(j + halfFilt) - k]*filter_sh[k + halfFilt];
    }
    rconvdata[rid*colsize + j] = tempsum;
    //rconvdata[rid*colsize + j] = rdata_sh[j + halfFilt];
}


__global__ void convRowGPU_comb(float *data, float *var, float *rowConvBig, float *rowConvSmall,
float *filterBig, float *filterSmall, int halfFiltBig, int halfFiltSmall, int colsize)
/*
convRowGPU
*/
{
    int k;
    int rid = blockIdx.x;// + threadIdx.y;
    int j = threadIdx.x;
    float tempsum = 0;

    //printf("threadIdx.y %d", threadIdx.y);

    volatile __shared__ float rdata_sh[275]; //must be padded image size + 1, or larger)
    //volatile __shared__ float rdata2_sh[275];
    __shared__ float filterBig_sh[12]; //must be at least the filter size
    __shared__ float filterSmall_sh[12]; //must be at least the filter size

    if (j < (halfFiltBig)){
        rdata_sh[j] = 0;
        rdata_sh[colsize + j + halfFiltBig] = 0;
        //printf("halfFiltBig, halfFiltSmall %d", halfFiltSmall);
    }
    rdata_sh[j + halfFiltBig] = data[rid*colsize + j]/var[rid*colsize + j];
    // Note that in the future, these next two lines could be combined by sending
    // in the filters concatenated into a single array
    if (j < (2*halfFiltBig)){
        filterBig_sh[j] = filterBig[j];
    }
    if (j < (2*halfFiltSmall)){
        filterSmall_sh[j] = filterSmall[j];
    }

    __syncthreads(); //make sure each thread is finished writing to shared memory

    //printf("threadIdx.y %d", threadIdx.y);
    for (k = -halfFiltBig; k < halfFiltBig; k++){
        tempsum += rdata_sh[(j + halfFiltBig) - k]*filterBig_sh[k + halfFiltBig];
    }
    rowConvBig[rid*colsize + j] = tempsum;
    //rowConvBig[rid*colsize + j] = rdata_sh[j + halfFiltBig][threadIdx.y];

    tempsum = 0;

    for (k = -halfFiltSmall; k <= halfFiltSmall; k++){
        tempsum += rdata_sh[(j + halfFiltBig) - k]*filterSmall_sh[k + halfFiltSmall];
    }
    rowConvSmall[rid*colsize + j] = tempsum;
    //rowConvSmall[rid*colsize + j] = rdata_sh[j + halfFiltBig][threadIdx.y];

}

__global__ void convRowGPU(float *data, float *var, float *rconvdata, float *gain, float *filter,// const int rowsize,
int halfFilt, const int colsize)
//
// This function takes input data and performs a row convolution. The convolution is stored in a separate
// output array.
//
{
    int k, halfFiltm1 = halfFilt-1;
    int rid = blockIdx.x;// + halfFilt;
    int j = threadIdx.x;// + halfFilt;
    float tempsum = 0;

    volatile __shared__ float rdata_sh[1075]; //should be changed to colsize (PADDED SIZE, or larger)
    __shared__ float filter_sh[12];
    //rdata_sh[j] = data[rid*colsize + j];


    if (j < (halfFilt)){
        rdata_sh[j] = 0;
        rdata_sh[colsize + j + halfFilt] = 0;
        //printf("colsize + halfFilt %d", (colsize + halfFilt));
    }
    rdata_sh[j + halfFilt] = data[rid*colsize + j]/(var[rid*colsize + j]*gain[rid*colsize + j]);
    if (j < (2*halfFilt)) filter_sh[j] = filter[j];

    //if (j > (colsize - 2*halfFilt)){
    //    filter_sh[j] = filter[j]; //trying to use other threads in case the two threads can run async.
    //    printf("%d,", j);
    //}

    __syncthreads();

    //if ((j==colsize-1)&&(rid==5)) printf("colsize + halfFilt %d", (colsize + j));

    for (k = -halfFilt; k <= halfFiltm1; k++){
        tempsum += rdata_sh[(j + halfFilt) - k]*filter_sh[k + halfFilt];
    }
    rconvdata[rid*colsize + j] = tempsum;
    //rconvdata[rid*colsize + j] = rdata_sh[j + halfFilt];
}


__global__ void convColGPU(float *data,  float *filter, int rowsize, int colsize, int halfFilt)
{
//
// convColGPU takes rov-convoled data as an input and performs a column-wise convolution. The
// completed 2D convolution is then stored in a separate output array.
//
    int k, halfFiltm1 = halfFilt-1;
    int cid = blockIdx.x;// + halfFilt;
    int j = threadIdx.x;// + halfFilt;
    float tempsum = 0;


    volatile __shared__ float cdata_sh[1075][1]; //should be changed to rowsize
    __shared__ float filter_sh[12];


    if (j < (halfFilt)){
        cdata_sh[j][0] = 0;
        cdata_sh[rowsize + j + halfFilt][0] = 0;
        //printf("colsize + halfFilt %d", (colsize + halfFilt));
    }
    cdata_sh[j + halfFilt][0] = data[j*colsize + cid];
    if (j < (2*halfFilt)) filter_sh[j] = filter[j];

    __syncthreads();


    for (k = -halfFilt; k <= halfFiltm1; k++){
        tempsum += cdata_sh[(j + halfFilt) - k][0]*filter_sh[k + halfFilt];
    }

    data[cid + j*colsize] = tempsum;
}

__global__ void smoothImGPU(float *uniflargedat, float *uniflargevar, float *unifsmalldat,  float *unifsmallvar,
int colsize, int halfFilt)
{
    int rid = blockIdx.x;
    int cid = threadIdx.x;
    int dloc = rid * colsize + cid;

    uniflargedat[dloc] = (unifsmalldat[dloc]/unifsmallvar[dloc]) - (uniflargedat[dloc]/uniflargevar[dloc]);

}


//__global__ void peakLocGPU(float *data, float *rconvdata, float *filter, const int rowsize,
//const int colsize, int halfFilt)