#include "hip/hip_runtime.h"
/*Compile the mex file: (old fashion). VS2008 version does not these steps.
 *
 * nvmex -f Z:\CUDA\nvmexoptsxp64.bat       GPUgaussMLE.cu -IC:\cuda\include -LC:\cuda\lib -lcudart
 * nvmex -f Z:\CUDA\nvmexopts_r2008a.bat    GPUgaussMLE.cu -IC:\cuda\include -LC:\cuda\lib -lcudart
 * nvmex -f Z:\CUDA\nvmexopts_9.bat    GPUgaussMLE.cu -IC:\cuda\include -LC:\cuda\lib -lcudart -output GPUgaussMLEfangMATLAB
 * nvmex -f Z:\cCode\CUDA\nvmexoptsXP32_VS2008_r2008.bat GPUgaussMLE.cu -IC:\cuda\include -LC:\cuda\lib -lcudart
 * nvmex -f Z:\cCode\CUDA\nvmexoptsxp64_VS2008SDK7.bat GPUgaussMLEf1.cu -IC:\cuda\include -LC:\cuda\lib64 -lcudart
 *
 * [Parameters CRLBs LL]=GPUgaussMLE(data,PSFSigma,iterations)
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include "mex.h"
#include "hip/hip_runtime.h"
#include "MatInvLib.h"
#include "GPUgaussLib.cuh"
#include "GPUgaussMLEv2.h"
// Thread block size
#define BSZ 64
//#define BSZ 64
#define MEM 1300
#define IMSZ 11
#define IMSZBIG 21
#define NK 128 //number of blocks to run in each kernel
#define pi 3.141592f
#define NV_P 4
#define NV_PS 5
#define NV_PZ 5
#define NV_PS2 6

#define NUM_VARS 4
#define BLOCK_MAX_SIZE 512

__device__ void kernel_sum_reduce(const int BlockSize, const int pixelIndex, volatile float *A, volatile float *Aout) {
    /* Calculate the sum by performing cascaded sums across threads

    */
    
    //unrolled loop to SUM across threads in O(log2(N))
    //first sum bottom half with top half,
    //then repeat with succesively smaller folds
    //concept borrowed/adapted from reduction code in cuda toolkit examples
    __syncthreads();

    //We expect our block size to be between around 120 and 441
    //pixels. First two rows are thus somewhat of a special case
    if ((BlockSize >= 256) && (pixelIndex < (BlockSize -256))){
        A[pixelIndex] += A[pixelIndex + 256];
    }

    __syncthreads();

    // This has to work as a 2nd iteration for a block size > 256 AND
    // as a first iteration for a block size between 128 and 256.
    if ((BlockSize >= 128) && (pixelIndex < min(128, BlockSize - 128))){
        A[pixelIndex] += A[pixelIndex + 128];
    }

    __syncthreads();

    if ((BlockSize >= 64) && (pixelIndex < 64)){
        A[pixelIndex] += A[pixelIndex + 64];
    }

    __syncthreads();

    //From here on in, we know that our BlockSize test is going to pass
    // so we can remove it

    //We're also just acting on one warp (<32 threads), so we a) don't need __syncthreads
    // and b) don't gain anything by further restricting the number of threads

    if ((pixelIndex < 32)){
        A[pixelIndex] += A[pixelIndex + 32];
        A[pixelIndex] += A[pixelIndex + 16];
        A[pixelIndex] += A[pixelIndex + 8];
        A[pixelIndex] += A[pixelIndex + 4];
        A[pixelIndex] += A[pixelIndex + 2];
    }

    //__syncthreads();

    // do the final sum and copy into output variable
    if (pixelIndex == 0){
        *Aout = (A[0] + A[1]);
    }

    __syncthreads();

    return;
}

__device__ void kernel_min_reduce(const int BlockSize, const int pixelIndex, volatile float *A, volatile float *Aout) {
    /* Calculate the minimum by performing cascaded sums across threads

    */
    
    //unrolled loop to take minumum across threads in O(log2(N))
    //first sum bottom half with top half,
    //then repeat with succesively smaller folds
    __syncthreads();

    //We expect our block size to be between around 120 and 441
    //pixels. First two rows are thus somewhat of a special case
    if ((BlockSize >= 256) && (pixelIndex < (BlockSize -256))){
        A[pixelIndex] = min(A[pixelIndex], A[pixelIndex + 256]);
    }

    __syncthreads();

    // This has to work as a 2nd iteration for a block size > 256 AND
    // as a first iteration for a block size between 128 and 256.
    if ((BlockSize >= 128) && (pixelIndex < min(128, BlockSize - 128))){
        A[pixelIndex] = min(A[pixelIndex], A[pixelIndex + 128]);
    }

    __syncthreads();

    if ((BlockSize >= 64) && (pixelIndex < 64)){
        A[pixelIndex] = min(A[pixelIndex], A[pixelIndex + 64]);
    }

    __syncthreads();

    //From here on in, we know that our BlockSize test is going to pass
    // so we can remove it

    //We're also just acting on one warp, so we a) don't need __syncthreads
    // and b) don't gain anything by further restricting the number of threads

    if ((pixelIndex < 32)){
        A[pixelIndex] = min(A[pixelIndex], A[pixelIndex + 32]);
        A[pixelIndex] = min(A[pixelIndex], A[pixelIndex + 16]);
        A[pixelIndex] = min(A[pixelIndex], A[pixelIndex + 8]);
        A[pixelIndex] = min(A[pixelIndex], A[pixelIndex + 4]);
        A[pixelIndex] = min(A[pixelIndex], A[pixelIndex + 2]);
    }

    //__syncthreads();

    if (pixelIndex == 0){
        *Aout = min(A[0], A[1]);
    }

    __syncthreads();

    return;
}

__device__ void kernel_max_reduce(const int BlockSize, const int pixelIndex, volatile float *A, volatile float *Aout) {
    /* Calculate the maximum by performing cascaded sums across threads

    */
    
    //unrolled loop to take max across threads in O(log(N))
    //first sum bottom half with top half,
    //then repeat with succesively smaller folds
    __syncthreads();

    //We expect our block size to be between around 120 and 441
    //pixels. First two rows are thus somewhat of a special case
    if ((BlockSize >= 256) && (pixelIndex < (BlockSize -256))){
        A[pixelIndex] = max(A[pixelIndex], A[pixelIndex + 256]);
    }

    __syncthreads();

    // This has to work as a 2nd iteration for a block size > 256 AND
    // as a first iteration for a block size between 128 and 256.
    if ((BlockSize >= 128) && (pixelIndex < min(128, BlockSize - 128))){
        A[pixelIndex] = max(A[pixelIndex], A[pixelIndex + 128]);
    }

    __syncthreads();

    if ((BlockSize >= 64) && (pixelIndex < 64)){
        A[pixelIndex] = max(A[pixelIndex], A[pixelIndex + 64]);
    }

    __syncthreads();

    //From here on in, we know that our BlockSize test is going to pass
    // so we can remove it

    //We're also just acting on one warp, so we a) don't need __syncthreads
    // and b) don't gain anything by further restricting the number of threads

    if ((pixelIndex < 32)){
        A[pixelIndex] = max(A[pixelIndex], A[pixelIndex + 32]);
        A[pixelIndex] = max(A[pixelIndex], A[pixelIndex + 16]);
        A[pixelIndex] = max(A[pixelIndex], A[pixelIndex + 8]);
        A[pixelIndex] = max(A[pixelIndex], A[pixelIndex + 4]);
        A[pixelIndex] = max(A[pixelIndex], A[pixelIndex + 2]);
    }

    //__syncthreads();

    if (pixelIndex == 0){
        *Aout = max(A[0], A[1]);
    }

    __syncthreads();

    return;
}

__device__ void kernel_sum_reduce_dual(const int BlockSize, const int pixelIndex, volatile float *A, volatile float *B, volatile float *Aout, volatile float *Bout) {
    /* Calculate the sum by performing cascaded sums across threads

    Sums two variables (A and B) at once to maximize throughput

    */

    //unrolled loop to SUM across threads in O(log2(N))
    //first sum bottom half with top half,
    //then repeat with succesively smaller folds
    __syncthreads();

    //We expect our block size to be between around 120 and 441
    //pixels. First two rows are thus somewhat of a special case
    if ((BlockSize >= 256) && (pixelIndex < (BlockSize -256))){
        A[pixelIndex] += A[pixelIndex + 256];
        B[pixelIndex] += B[pixelIndex + 256];
    }

    __syncthreads();

    // This has to work as a 2nd iteration for a block size > 256 AND
    // as a first iteration for a block size between 128 and 256.
    if ((BlockSize >= 128) && (pixelIndex < min(128, BlockSize - 128))){
        A[pixelIndex] += A[pixelIndex + 128];
        B[pixelIndex] += B[pixelIndex + 128];
    }

    __syncthreads();

    if ((BlockSize >= 64) && (pixelIndex < 64)){
        A[pixelIndex] += A[pixelIndex + 64];
        B[pixelIndex] += B[pixelIndex + 64];
    }

    __syncthreads();

    //From here on in, we know that our BlockSize test is going to pass
    // so we can remove it

    //We're also just acting on one warp, so we a) don't need __syncthreads
    // and b) don't gain anything by further restricting the number of threads

    if ((pixelIndex < 32)){
        A[pixelIndex] += A[pixelIndex + 32];
        B[pixelIndex] += B[pixelIndex + 32];

        A[pixelIndex] += A[pixelIndex + 16];
        B[pixelIndex] += B[pixelIndex + 16];
    
        A[pixelIndex] += A[pixelIndex + 8];
        B[pixelIndex] += B[pixelIndex + 8];
    
        A[pixelIndex] += A[pixelIndex + 4];
        B[pixelIndex] += B[pixelIndex + 4];
    
        A[pixelIndex] += A[pixelIndex + 2];
        B[pixelIndex] += B[pixelIndex + 2];
    }

    //__syncthreads();

    if (pixelIndex == 0){
        *Aout = (A[0] + A[1]);
        *Bout = (B[0] + B[1]);
    }

    __syncthreads();

    return;
}


__global__ void kernel_MLEFit_pix_threads(float *d_data, float PSFSigma, int sz, int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int Nfits, float *d_varim, float *d_gainim, int calcCRB){
    /* A version of MLEFit that uses per-pixel, rather than per fit threads 

    Each block consists corresponds to one ROI. threadIdx.x is the x pixel coordinate, 
    threadIdx.y is the y coordinate.

    David Baddeley
    Feb 2016

    */
    

    //these will store the data for the pixel being processed
    float pixel_data;
    float pixel_variance;
    float pixel_gain;
    float pixel_variance_over_gain_squared;

    //allocate arrays for matrices
    __shared__ float M[NUM_VARS*NUM_VARS];
    __shared__ float Diag[NUM_VARS];
    __shared__ float Minv[NUM_VARS*NUM_VARS];
    
    //One block = one ROI
    int BlockSize = blockDim.x*blockDim.y;
    //#define BlockSize (blockDim.x*blockDim.y)

    int blockOffset = BlockSize*blockIdx.x; //the offset in memory to the current ROI
    int pixelIndex = threadIdx.x + blockDim.x*threadIdx.y; //index of pixel in ROI

    //printf("pixelIndex: %d\n", pixelIndex);

    /*if (pixelIndex == 0){
        printf("blockIdx.x: %d\n", blockIdx.x);
        printf("BlockSize: %d\n", BlockSize);
        printf("blockOffset: %d\n", blockOffset);
    }*/
    
    //float gain=0.45;
    //int ii, jj, kk, ll;
    int kk, ll;
    float model, cf, df, data;
    float tmp;
    float PSFy, PSFx;

    volatile __shared__ float data_sum;

    //Allocate arrays for the gradients etc ...
    //float dudt[NUM_VARS];
    //float d2udt2[NUM_VARS];

    __shared__ float dudt[NUM_VARS];
    __shared__ float d2udt2[NUM_VARS];

    //two temporary arrays which are used whenever we want to reduce (sum, min, max)
    //across threads
    volatile __shared__ float s_temp0[BLOCK_MAX_SIZE];
    volatile __shared__ float s_temp1[BLOCK_MAX_SIZE];
    
    //The numerator and denominator for the update step. These are used from multiple threads, and need
    //to be shared
    volatile __shared__ float NR_Numerator[NUM_VARS];
    volatile __shared__ float NR_Denominator[NUM_VARS];

    //the parameter vector
    volatile __shared__ float theta[NUM_VARS];

    //can these be const??? I don't see maxjump being revised anywhere
    const float maxjump[NUM_VARS]={1e0, 1e0, 1e2, 2e0}; // maxjump is revised such that the intensity can converge to the correct value
    const float gamma[NUM_VARS]={1.0, 1.0, 0.5, 1.0};
    
    //variable to store maximum intensity in ROI
    //float Nmax;

    //Prevent read/write past end of array
    //if ((bx*BlockSize+tx)>=Nfits) return;
    
    //zero arrays (each thread zeros one location)
    // As we have more pixels than entries in the matrix, we will only use the first NV*NV threads
    // We've effectively parallized the loop across the threads
    // CHECK - Do we still need to zero M (we now do the sum and then assign, rather than a progressive sum)
    if (pixelIndex < NUM_VARS*NUM_VARS){
        M[pixelIndex] = 0;
        Minv[pixelIndex] = 0;
    }
    

    //maybe put a __syncthreads here to ensure following ops are coallesced??
    __syncthreads(); //Is this necessary?

    //load data from global memory (each thread loads the data for it's pixel)
    //as each thread accesses a consecutive memory location, access should be coalesced
    pixel_data = d_data[blockOffset + pixelIndex];
    pixel_variance =(d_varim[blockOffset + pixelIndex]);
    pixel_gain = (d_gainim[blockOffset + pixelIndex]);
    
    //Precompute these quantities (previously re-evaluated on each iteration)
    pixel_variance_over_gain_squared = pixel_variance/(pixel_gain*pixel_gain);
    data=pixel_data + pixel_variance_over_gain_squared;

    //wait untill all loads have taken place
    __syncthreads();

    //initial values
    //==============     
    //Find Center of mass

    //find sum of all pixels in ROI
    s_temp0[pixelIndex] = pixel_data;
    kernel_sum_reduce(BlockSize, pixelIndex, s_temp0, &data_sum);

    //printf("data_sum: %f", data_sum);

    // Find centroid
    s_temp0[pixelIndex] = pixel_data*threadIdx.x/data_sum;
    s_temp1[pixelIndex] = pixel_data*threadIdx.y/data_sum;

    kernel_sum_reduce_dual(BlockSize, pixelIndex, s_temp0, s_temp1, &theta[0], &theta[1]);

    /*if (pixelIndex == 0){
        printf("data_sum: %f\n", data_sum);
        printf("x0, y0: %f, %f\n", theta[0], theta[1]);
    }*/

    //find max and min values
    s_temp0[pixelIndex] = pixel_data;
    s_temp1[pixelIndex] = pixel_data;

    kernel_min_reduce(BlockSize, pixelIndex, s_temp0, &theta[3]);
    kernel_max_reduce(BlockSize, pixelIndex, s_temp1, &tmp);
    
    if (pixelIndex == 0)
        //We only need to do this once - do it in thread 0
        theta[2]=max(0.0, (tmp-theta[3])*2*pi*PSFSigma*PSFSigma);

    __syncthreads();
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop

        PSFx=kernel_IntGauss1D(threadIdx.x, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(threadIdx.y, theta[1], PSFSigma);
            
        model=theta[3] + theta[2]*PSFx*PSFy + pixel_variance_over_gain_squared;
        //data=pixel_data + pixel_variance_over_gain_squared; //now calculated outside loop
            
        //calculating derivatives
        kernel_DerivativeIntGauss1D(threadIdx.x, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], &d2udt2[0]);
        kernel_DerivativeIntGauss1D(threadIdx.y, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], &d2udt2[1]);
        dudt[2] = PSFx*PSFy;
        d2udt2[2] = 0.0;
        dudt[3] = 1.0;
        d2udt2[3] = 0.0;
            
        cf=0.0;
        df=0.0;
        if (model>10e-3) cf=data/model-1;
        if (model>10e-3) df=data/(model*model);
        cf=min(cf, 10e4);
        df=min(df, 10e4);
            
        for (ll=0;ll < NUM_VARS;ll++){
            s_temp0[pixelIndex] = dudt[ll]*cf;
            s_temp1[pixelIndex] = d2udt2[ll]*cf - pow(dudt[ll], 2)*df;

            //sum over all pixels
            kernel_sum_reduce_dual(BlockSize, pixelIndex, s_temp0, s_temp1, &NR_Numerator[ll], &NR_Denominator[ll]);
        }

        
        // The update
        ll = pixelIndex;
        //rather than looping over the variables, use the first NUM_VARS threads
        if (ll < NUM_VARS){
            if (kk<2)
                theta[ll]-=gamma[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
            else
                theta[ll]-=min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);

        }
        __syncthreads();
        
        if (pixelIndex == 0){
            // Do this in the 0th thread only - i.e. once per block/ROI
            // Any other constraints
            theta[2]=max(theta[2], 1.0);
            theta[3]=max(theta[3], 0.01);

            /*printf("block: %d, iteration: %d, theta: [%f, %f, %f, %f]\n", blockIdx.x, kk, theta[0], theta[1], theta[2], theta[3]);
            printf("block: %d, iteration: %d, num/dem: [%f/%f, %f/%f, %f/%f, %f/%f]\n", blockIdx.x, kk, 
                NR_Numerator[0], NR_Denominator[0], NR_Numerator[1], NR_Denominator[1],
                NR_Numerator[2], NR_Denominator[2], NR_Numerator[3], NR_Denominator[3] 
                );*/
        }

        __syncthreads();
        
    }
    
    if (calcCRB){
        // Calculating the CRLB and LogLikelihood
        //Div=0.0;
            
        //evaluate model with final parameters
        //Note that if fit is converged, we could probably skip this step
        //as model, dudt etc ... should still have the correct values from the last fit iteration
        PSFx=kernel_IntGauss1D(threadIdx.x, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(threadIdx.y, theta[1], PSFSigma);
            
        model=theta[3] + theta[2]*PSFx*PSFy + pixel_variance_over_gain_squared;
            
        //calculating derivatives
        kernel_DerivativeIntGauss1D(threadIdx.x, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(threadIdx.y, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], NULL);
        
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0;
            
        //Building the Fisher Information Matrix
        for (kk=0;kk<NUM_VARS;kk++) for (ll=kk;ll<NUM_VARS;ll++){

            //reuse s_temp0 here to save memory
            s_temp0[pixelIndex] = dudt[ll]*dudt[kk]/model;

            //sum over all pixels
            kernel_sum_reduce(BlockSize, pixelIndex, s_temp0, &M[kk*NUM_VARS+ll]);

            if (pixelIndex == 0){
                //execute in first thread of block only

                //exploit known matrix symetry and copy value to the mirrored entry
                M[ll*NUM_VARS+kk] = M[kk*NUM_VARS+ll];
            }

            __syncthreads();
            
        }
        
        //LogLikelyhood
        if (model>0){
            if (data>0){
                //reuse s_temp0 here to save memory
                s_temp0[pixelIndex] = data*log(model)-model-data*log(data)+data;
            } //Div+=data*log(model)-model-data*log(data)+data;
            else {
                s_temp0[pixelIndex] = -model;
            } //Div+=-model;
        } else s_temp0[pixelIndex] =0;

        //sum over all pixels
        kernel_sum_reduce(BlockSize, pixelIndex, s_temp0, &tmp);

        if (pixelIndex == 0){
            //only in first thread ...

            //write our log likelihood back into global memory
            d_LogLikelihood[blockIdx.x] = tmp;
        
            // Matrix inverse (CRLB=F^-1) and output assigments 
            kernel_MatInvN(M, Minv, Diag, NUM_VARS);
        }

    }
    __syncthreads();
    
    //write to global arrays
    //CHECK!! - are output arrays transposed???
    //We want this matrix orientation as it optimizes memory access speed (coalescing) 
    if (pixelIndex < NUM_VARS){
        d_Parameters[NUM_VARS*blockIdx.x + pixelIndex] = theta[pixelIndex];
        d_CRLBs[NUM_VARS*blockIdx.x + pixelIndex] = Diag[pixelIndex];
    }
    
    return;
}

#define NUM_VARS_ASTIG 6

__global__ void kernel_MLEFit_pix_threads_astig(float *d_data, float PSFSigma, int sz, int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int Nfits, float *d_varim, float *d_gainim, int calcCRB){
    /* A version of MLEFit that uses per-pixel, rather than per fit threads 

    Each block consists corresponds to one ROI. threadIdx.x is the x pixel coordinate, 
    threadIdx.y is the y coordinate.

    David Baddeley
    Feb 2016

    */
    

    //these will store the data for the pixel being processed
    float pixel_data;
    float pixel_variance;
    float pixel_gain;
    float pixel_variance_over_gain_squared;

    //allocate arrays for matrices
    float M[NUM_VARS_ASTIG*NUM_VARS_ASTIG];
    __shared__ float Diag[NUM_VARS_ASTIG];
    float Minv[NUM_VARS_ASTIG*NUM_VARS_ASTIG];
    
    //One block = one ROI
    int BlockSize = blockDim.x*blockDim.y;
    int blockOffset = BlockSize*blockIdx.x; //the offset in memory to the current ROI
    int pixelIndex = threadIdx.x + blockDim.x*threadIdx.y; //index of pixel in ROI

    //printf("pixelIndex: %d\n", pixelIndex);

    /*if (pixelIndex == 0){
        printf("blockIdx.x: %d\n", blockIdx.x);
        printf("BlockSize: %d\n", BlockSize);
        printf("blockOffset: %d\n", blockOffset);
    }*/
    
    //float gain=0.45;
    //int ii, jj, kk, ll;
    int kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;

    volatile __shared__ float data_sum;

    //Allocate arrays for the gradients etc ...
    float dudt[NUM_VARS_ASTIG];
    float d2udt2[NUM_VARS_ASTIG];

    //two temporary arrays which are used whenever we want to reduce (sum, min, max)
    //across threads
    volatile __shared__ float s_temp0[BLOCK_MAX_SIZE];
    volatile __shared__ float s_temp1[BLOCK_MAX_SIZE];
    
    //The numerator and denominator for the update step. These are used from multiple threads, and need
    //to be shared
    volatile __shared__ float NR_Numerator[NUM_VARS_ASTIG];
    volatile __shared__ float NR_Denominator[NUM_VARS_ASTIG];

    //the parameter vector
    volatile __shared__ float theta[NUM_VARS_ASTIG];

    //can these be const??? I don't see maxjump being revised anywhere
    const float maxjump[NUM_VARS_ASTIG]={1e0, 1e0, 1e2, 2e0, 1e-1, 1e-1}; // maxjump is revised such that the intensity can converge to the correct value
    const float gamma[NUM_VARS_ASTIG]={1.0, 1.0, 0.5, 1.0, 1.0, 1.0};
    
    //variable to store maximum intensity in ROI
    float Nmax;

    //Prevent read/write past end of array
    //if ((bx*BlockSize+tx)>=Nfits) return;
    
    //zero arrays (each thread zeros one location)
    // As we have more pixels than entries in the matrix, we will only use the first NV*NV threads
    // We've effectively parallized the loop across the threads
    // CHECK - Do we still need to zero M (we now do the sum and then assign, rather than a progressive sum)
    if (pixelIndex < NUM_VARS_ASTIG*NUM_VARS_ASTIG){
        M[pixelIndex] = 0;
        Minv[pixelIndex] = 0;
    }
    

    //maybe put a __syncthreads here to ensure following ops are coallesced??
    __syncthreads(); //Is this necessary?

    //load data from global memory (each thread loads the data for it's pixel)
    //as each thread accesses a consecutive memory location, access should be coalesced
    pixel_data = d_data[blockOffset + pixelIndex];
    pixel_variance = d_varim[blockOffset + pixelIndex];
    pixel_gain = d_gainim[blockOffset + pixelIndex];
    
    //Precompute these quantities (previously re-evaluated on each iteration)
    pixel_variance_over_gain_squared = pixel_variance/(pixel_gain*pixel_gain);
    data=pixel_data + pixel_variance_over_gain_squared;

    //wait untill all loads have taken place
    __syncthreads();

    //initial values
    //==============     
    //Find Center of mass

    //find sum of all pixels in ROI
    s_temp0[pixelIndex] = pixel_data;
    kernel_sum_reduce(BlockSize, pixelIndex, s_temp0, &data_sum);

    //printf("data_sum: %f", data_sum);

    // Find centroid
    s_temp0[pixelIndex] = pixel_data*threadIdx.x/data_sum;
    s_temp1[pixelIndex] = pixel_data*threadIdx.y/data_sum;

    kernel_sum_reduce_dual(BlockSize, pixelIndex, s_temp0, s_temp1, &theta[0], &theta[1]);

    /*if (pixelIndex == 0){
        printf("data_sum: %f\n", data_sum);
        printf("x0, y0: %f, %f\n", theta[0], theta[1]);
    }*/

    //find max and min values
    s_temp0[pixelIndex] = pixel_data;
    s_temp1[pixelIndex] = pixel_data;

    kernel_min_reduce(BlockSize, pixelIndex, s_temp0, &theta[3]);
    kernel_max_reduce(BlockSize, pixelIndex, s_temp1, &Nmax);
    
    if (pixelIndex == 0)
        //We only need to do this once - do it in thread 0
        theta[2]=max(0.0, (Nmax-theta[3])*2*pi*PSFSigma*PSFSigma);

        theta[4]=PSFSigma;
        theta[5]=PSFSigma;

    __syncthreads();
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop

        PSFx=kernel_IntGauss1D(threadIdx.x, theta[0], theta[4]);
        PSFy=kernel_IntGauss1D(threadIdx.y, theta[1], theta[5]);
            
        model=theta[3] + theta[2]*PSFx*PSFy + pixel_variance_over_gain_squared;
        //data=pixel_data + pixel_variance_over_gain_squared; //now calculated outside loop
            
        //calculating derivatives
        kernel_DerivativeIntGauss1D(threadIdx.x, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
        kernel_DerivativeIntGauss1D(threadIdx.y, theta[1], theta[5], theta[2], PSFx, &dudt[1], &d2udt2[1]);
        kernel_DerivativeIntGauss1DSigma(threadIdx.x, theta[0], theta[4], theta[2], PSFy, &dudt[4], &d2udt2[4]);
        kernel_DerivativeIntGauss1DSigma(threadIdx.y, theta[1], theta[5], theta[2], PSFx, &dudt[5], &d2udt2[5]);
        dudt[2] = PSFx*PSFy;
        d2udt2[2] = 0.0;
        dudt[3] = 1.0;
        d2udt2[3] = 0.0;
            
        cf=0.0;
        df=0.0;
        if (model>10e-3) cf=data/model-1;
        if (model>10e-3) df=data/(model*model);
        cf=min(cf, 10e4);
        df=min(df, 10e4);
            
        for (ll=0;ll < NUM_VARS_ASTIG;ll++){
            s_temp0[pixelIndex] = dudt[ll]*cf;
            s_temp1[pixelIndex] = d2udt2[ll]*cf - pow(dudt[ll], 2)*df;

            //sum over all pixels
            kernel_sum_reduce_dual(BlockSize, pixelIndex, s_temp0, s_temp1, &NR_Numerator[ll], &NR_Denominator[ll]);
        }

        
        // The update
        ll = pixelIndex;
        //rather than looping over the variables, use the first NUM_VARS_ASTIG threads
        if (ll < NUM_VARS_ASTIG){
            if (kk<2)
                theta[ll]-=gamma[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
            else
                theta[ll]-=min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);

        }
        __syncthreads();
        
        if (pixelIndex == 0){
            // Do this in the 0th thread only - i.e. once per block/ROI
            // Any other constraints
            theta[2]=max(theta[2], 1.0);
            theta[3]=max(theta[3], 0.01);
            theta[4]=max(theta[4], PSFSigma/10.0);
            theta[5]=max(theta[5], PSFSigma/10.0); 

            /*printf("block: %d, iteration: %d, theta: [%f, %f, %f, %f]\n", blockIdx.x, kk, theta[0], theta[1], theta[2], theta[3]);
            printf("block: %d, iteration: %d, num/dem: [%f/%f, %f/%f, %f/%f, %f/%f]\n", blockIdx.x, kk, 
                NR_Numerator[0], NR_Denominator[0], NR_Numerator[1], NR_Denominator[1],
                NR_Numerator[2], NR_Denominator[2], NR_Numerator[3], NR_Denominator[3] 
                );*/
        }

        __syncthreads();
        
    }
    
    if (calcCRB){
        // Calculating the CRLB and LogLikelihood
        //Div=0.0;
            
        //evaluate model with final parameters
        //Note that if fit is converged, we could probably skip this step
        //as model, dudt etc ... should still have the correct values from the last fit iteration
        PSFx=kernel_IntGauss1D(threadIdx.x, theta[0], theta[4]);
        PSFy=kernel_IntGauss1D(threadIdx.y, theta[1], theta[5]);
            
        model=theta[3] + theta[2]*PSFx*PSFy + pixel_variance_over_gain_squared;
            
        //calculating derivatives
        kernel_DerivativeIntGauss1D(threadIdx.x, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(threadIdx.y, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss1DSigma(threadIdx.x, theta[0], theta[4], theta[2], PSFy, &dudt[4], NULL);
        kernel_DerivativeIntGauss1DSigma(threadIdx.y, theta[1], theta[5], theta[2], PSFx, &dudt[5], NULL);
        
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0;
            
        //Building the Fisher Information Matrix
        for (kk=0;kk<NUM_VARS_ASTIG;kk++) for (ll=kk;ll<NUM_VARS_ASTIG;ll++){

            //reuse s_temp0 here to save memory
            s_temp0[pixelIndex] = dudt[ll]*dudt[kk]/model;

            //sum over all pixels
            kernel_sum_reduce(BlockSize, pixelIndex, s_temp0, &M[kk*NUM_VARS_ASTIG+ll]);

            if (pixelIndex == 0){
                //execute in first thread of block only

                //exploit known matrix symetry and copy value to the mirrored entry
                M[ll*NUM_VARS_ASTIG+kk] = M[kk*NUM_VARS_ASTIG+ll];
            }

            __syncthreads();
            
        }
        
        //LogLikelyhood
        if (model>0){
            if (data>0){
                //reuse s_temp0 here to save memory
                s_temp0[pixelIndex] = data*log(model)-model-data*log(data)+data;
            } //Div+=data*log(model)-model-data*log(data)+data;
            else {
                s_temp0[pixelIndex] = -model;
            } //Div+=-model;
        } else s_temp0[pixelIndex] =0;

        //sum over all pixels
        kernel_sum_reduce(BlockSize, pixelIndex, s_temp0, &Div);

        if (pixelIndex == 0){
            //only in first thread ...

            //write our log likelihood back into global memory
            d_LogLikelihood[blockIdx.x] = Div;
        
            // Matrix inverse (CRLB=F^-1) and output assigments 
            kernel_MatInvN(M, Minv, Diag, NUM_VARS_ASTIG);
        }

    }
    __syncthreads();
    
    //write to global arrays
    //CHECK!! - are output arrays transposed???
    //We want this matrix orientation as it optimizes memory access speed (coalescing) 
    if (pixelIndex < NUM_VARS_ASTIG){
        d_Parameters[NUM_VARS_ASTIG*blockIdx.x + pixelIndex] = theta[pixelIndex];
        d_CRLBs[NUM_VARS_ASTIG*blockIdx.x + pixelIndex] = Diag[pixelIndex];
    }
    
    return;
}



//theta is: {x,y,N,bg}
__global__ void kernel_MLEFit(float *d_data, float PSFSigma, int sz, int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int Nfits, float *d_varim, float *d_gainim){
    
    __shared__ float s_data[MEM];
	__shared__ float s_varim[MEM];
	__shared__ float s_gainim[MEM];
    float M[NV_P*NV_P], Diag[NV_P], Minv[NV_P*NV_P];
    int tx = threadIdx.x;
    int bx = blockIdx.x;

    int BlockSize = blockDim.x; //Number of threads in current block 

	//float gain=0.45;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_P;
    float dudt[NV_P];
    float d2udt2[NV_P];
    float NR_Numerator[NV_P], NR_Denominator[NV_P];
    float theta[NV_P];
    float maxjump[NV_P]={1e0, 1e0, 1e2, 2e0}; // maxjump is revised such that the intensity can converge to the correct value
    float gamma[NV_P]={1.0, 1.0, 0.5, 1.0};
    float Nmax;

    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
    for (ii=0;ii<NV*NV;ii++)M[ii]=0;
    for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;
    //load data
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++)
	{
        /*s_data[sz*sz*tx+sz*jj+ii]=d_data[sz*sz*bx*BlockSize+sz*sz*tx+sz*jj+ii];
		s_varim[sz*sz*tx+sz*jj+ii]=d_varim[sz*sz*bx*BlockSize+sz*sz*tx+sz*jj+ii];
		s_gainim[sz*sz*tx+sz*jj+ii]=d_gainim[sz*sz*bx*BlockSize+sz*sz*tx+sz*jj+ii];*/

        s_data[sz*sz*tx + sz*jj + ii]=d_data[sz*sz*bx*BlockSize + sz*sz*tx + sz*jj + ii];
        s_varim[sz*sz*tx + sz*jj + ii]=d_varim[sz*sz*bx*BlockSize + sz*sz*tx + sz*jj+ii];
        s_gainim[sz*sz*tx + sz*jj + ii]=d_gainim[sz*sz*bx*BlockSize + sz*sz*tx + sz*jj+ii];
	}
    //initial values
    kernel_CenterofMass2D(sz, &s_data[sz*sz*tx], &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, &s_data[sz*sz*tx], &Nmax, &theta[3]);
    theta[2]=max(0.0, (Nmax-theta[3])*2*pi*PSFSigma*PSFSigma);
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
        for (ll=0;ll<NV;ll++){
            NR_Numerator[ll]=0;
            NR_Denominator[ll]=0;}
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
            PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
            
            model=theta[3]+theta[2]*PSFx*PSFy+s_varim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii];
            data=s_data[sz*sz*tx+sz*jj+ii]+s_varim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], &d2udt2[1]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0;
            dudt[3] = 1.0;
            d2udt2[3] = 0.0;
            
            cf=0.0;
            df=0.0;
            if (model>10e-3) cf=data/model-1;
            if (model>10e-3) df=data/pow(model, 2);
            cf=min(cf, 10e4);
            df=min(df, 10e4);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update
        if (kk<2)
            for (ll=0;ll<NV;ll++)
                theta[ll]-=gamma[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        else
            for (ll=0;ll<NV;ll++)
                theta[ll]-=min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0);
        theta[3]=max(theta[3], 0.01);
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
        
        model=theta[3]+theta[2]*PSFx*PSFy+s_varim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii];
        data=s_data[sz*sz*tx+sz*jj+ii]+s_varim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk + BlockSize*bx + tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk + BlockSize*bx + tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx + tx] = Div;
	//d_LogLikelihood[BlockSize*bx+tx] = 1;
    
    return;
}

__global__ void kernel_MLEFit_sigma(float *d_data, float PSFSigma, int sz, int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int Nfits, float *d_varim, float *d_gainim){
    
    __shared__ float s_data[MEM];
	__shared__ float s_varim[MEM];
	__shared__ float s_gainim[MEM];
    float M[NV_PS*NV_PS], Diag[NV_PS], Minv[NV_PS*NV_PS];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
	//float gain=0.45;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_PS;
    float dudt[NV_PS];
    float d2udt2[NV_PS];
    float NR_Numerator[NV_PS], NR_Denominator[NV_PS];
    float theta[NV_PS];
    float maxjump[NV_PS]={1e0, 1e0, 1e2, 2e0, 5e-1};
    float gamma[NV_PS]={1.0, 1.0, 0.5, 1.0, 1.0};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
    for (ii=0;ii<NV*NV;ii++)M[ii]=0;
    for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;
      
    //copy in data
	for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        s_data[sz*sz*tx+sz*jj+ii]=d_data[sz*sz*bx*BlockSize+sz*sz*tx+sz*jj+ii];
		s_varim[sz*sz*tx+sz*jj+ii]=d_varim[sz*sz*bx*BlockSize+sz*sz*tx+sz*jj+ii];
		s_gainim[sz*sz*tx+sz*jj+ii]=d_gainim[sz*sz*bx*BlockSize+sz*sz*tx+sz*jj+ii];
	}
    
    //initial values
    kernel_CenterofMass2D(sz, &s_data[sz*sz*tx], &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, &s_data[sz*sz*tx], &Nmax, &theta[3]);
    theta[2]=max(0.0, (Nmax-theta[3])*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
        for (ll=0;ll<NV;ll++){
            NR_Numerator[ll]=0;
            NR_Denominator[ll]=0;}
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[4]);
            
            model=theta[3]+theta[2]*PSFx*PSFy+s_varim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii];
            data=s_data[sz*sz*tx+sz*jj+ii]+s_varim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], &d2udt2[4]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0;
            dudt[3] = 1.0;
            d2udt2[3] = 0.0;
            
            cf=0.0;
            df=0.0;
            if (model>10e-3) cf=data/model-1;
            if (model>10e-3) df=data/pow(model, 2);
            cf=min(cf, 10e4);
            df=min(df, 10e4);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update
        if (kk<5)
            for (ll=0;ll<NV;ll++)
                theta[ll]-=gamma[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        else
            for (ll=0;ll<NV;ll++)
                theta[ll]-=min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0);
        theta[3]=max(theta[3], 0.01);
        theta[4]=max(theta[4], 0.5);
        theta[4]=min(theta[4], sz/2.0);
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
        
        model=theta[3]+theta[2]*PSFx*PSFy+s_varim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii];
        data=s_data[sz*sz*tx+sz*jj+ii]+s_varim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii]/s_gainim[sz*sz*tx+sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
  
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
	//d_LogLikelihood[BlockSize*bx+tx] = 1;
    
    
    return;
}

//******************* 3D localization using z fit does not work for this version of sCMOS software **********************//
__global__ void kernel_MLEFit_z(float *d_data, float PSFSigma_x, float Ax, float Ay, float Bx, float By, float gamma, float d, float PSFSigma_y, int sz, int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int Nfits, float *d_varim){
    __shared__ float s_data[MEM];
    float M[5*5], Diag[5], Minv[5*5];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=5;
    float dudt[5];
    float d2udt2[5];
    float NR_Numerator[5], NR_Denominator[5];
    float theta[5];
    float maxjump[5]={1e0, 1e0, 1e2, 2e0, 1e-1};
    float g[5]={1.0, 1.0, 0.5, 1.0, 1.0};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;

    for (ii=0;ii<NV*NV;ii++)M[ii]=0;
    for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;
    
    //copy in data
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++)
        s_data[sz*sz*tx+sz*jj+ii]=d_data[sz*sz*bx*BlockSize+sz*sz*tx+sz*jj+ii];
    
    //initial values
    kernel_CenterofMass2D(sz, &s_data[sz*sz*tx], &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma_x, &s_data[sz*sz*tx], &Nmax, &theta[3]);
    theta[2]=max(0.0, (Nmax-theta[3])*2*pi*PSFSigma_x*PSFSigma_y*sqrt(2.0f));
    theta[4]=0;
   
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
        for (ll=0;ll<NV;ll++){
            NR_Numerator[ll]=0;
            NR_Denominator[ll]=0;}
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, dudt, d2udt2);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*sz*tx+sz*jj+ii];
            
            //calculating remaining derivatives
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0;
            dudt[3] = 1.0;
            d2udt2[3] = 0.0;
            
            cf=0.0;
            df=0.0;
            if (model>10e-3) cf=data/model-1;
            if (model>10e-3) df=data/pow(model, 2);
            cf=min(cf, 10e4);
            df=min(df, 10e4);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update
        if (kk<2)
            for (ll=0;ll<NV;ll++)
                theta[ll]-=g[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        else
            for (ll=0;ll<NV;ll++)
                theta[ll]-=min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0);
        theta[3]=max(theta[3], 0.01);
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay, Bx,By, gamma, d, &PSFx, &PSFy, dudt, NULL);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*sz*tx+sz*jj+ii];
        
        //calculating remaining derivatives
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0;
       
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) 
    kernel_MatInvN(M, Minv, Diag, NV);
  
   //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}

//************* sigma fit does not work for this version of sCMOS software***********************************//
__global__ void kernel_MLEFit_sigmaxy(float *d_data, float PSFSigma, int sz, int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int Nfits, float *d_varim){
 
    __shared__ float s_data[MEM]; //note doesn't allocate memory for gain or variance
    float M[6*6], Diag[6], Minv[6*6];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=6;
    float dudt[6];
    float d2udt2[6];
    float NR_Numerator[6], NR_Denominator[6];
    float theta[6];
    float maxjump[6]={1e0, 1e0, 1e2, 2e0, 1e-1,1e-1};
    float g[6]={1.0, 1.0, 0.5, 1.0, 1.0,1.0};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
    for (ii=0;ii<NV*NV;ii++)M[ii]=0;
    for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;
    
    //copy in data
    
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++)
        s_data[sz*sz*tx+sz*jj+ii]=d_data[sz*sz*bx*BlockSize+sz*sz*tx+sz*jj+ii];
    
    //initial values
    kernel_CenterofMass2D(sz, &s_data[sz*sz*tx], &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, &s_data[sz*sz*tx], &Nmax, &theta[3]);
    theta[2]=max(0.0, (Nmax-theta[3])*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    theta[5]=PSFSigma;
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
        for (ll=0;ll<NV;ll++){
            NR_Numerator[ll]=0;
            NR_Denominator[ll]=0;}
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*sz*tx+sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], &d2udt2[4]);
            kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], &d2udt2[5]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0;
            dudt[3] = 1.0;
            d2udt2[3] = 0.0;
            
            cf=0.0;
            df=0.0;
            if (model>10e-3) cf=data/model-1;
            if (model>10e-3) df=data/pow(model, 2);
            cf=min(cf, 10e4);
            df=min(df, 10e4);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update
            for (ll=0;ll<NV;ll++)
                theta[ll]-=g[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0);
        theta[3]=max(theta[3], 0.01);
        theta[4]=max(theta[4], PSFSigma/10.0);
        theta[5]=max(theta[5], PSFSigma/10.0);  
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
        PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*sz*tx+sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], NULL);
        kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
   
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}
