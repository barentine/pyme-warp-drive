#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "definitions.h"
#include "kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void kernel_example(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
